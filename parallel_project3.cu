
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <chrono>
using namespace std;

  int max_rows = 1000;
  int max_col = 1000;
  int max_color = 255;
  #define N 100000
class Point
{
public:
  Point(int x, int y, int r, int g, int b);
  int getx();
  int gety();
  vector<int> getRGB();
  void setRGB(vector<int> rgb);
  void setShortestDistance(int s_distance);
  int getShortestDistance();
  int x;
  int y;
  int r;
  int g;
  int b;
  int shortest_distance = -1;
};

void Point::setRGB(vector<int> rgb){

  this->r = rgb.at(0);
  this->g = rgb.at(1);
  this->b = rgb.at(2);
}
Point::Point(int x, int y, int r, int g, int b)
{
  this->x = x;
  this->y = y;
  this->r = r;
  this->g = g;
  this->b = b;
  this->shortest_distance = max_col * max_rows;
}
void Point::setShortestDistance(int s_distance){
  
  this->shortest_distance = s_distance;
}
int Point::getx()
{
  return this->x;
}
int Point::gety()
{
  return this->y;
}
int Point::getShortestDistance(){

  return this->shortest_distance;
}
vector<int> Point::getRGB()
{

  vector<int> color;
  color.push_back(this->r);
  color.push_back(this->g);
  color.push_back(this->b);

  return color;
}

void calculate_distanace_from_pixels_to_seeds(vector<Point> pixels, vector<Point> seeds_point);

__device__ int counter =0 ;
__global__
void calculate_distanace_from_pixels_to_seeds(Point *pixels, Point *seeds_point, int seed, int max_rows, int max_col, int *total_pixels){

  int thread = threadIdx.x;

while ( counter < max_rows * max_col -1)
{
  int a = atomicAdd(&counter, 1);
  for (int j = 0; j < seed; j++){

    int x = pixels[a].x - seeds_point[j].x;
    x = x*x;

    int y = pixels[a].y - seeds_point[j].y;
    y = y*y;
    int distance = sqrtf(x + y);

    if(j == 0){
      pixels[a].shortest_distance = distance;

      pixels[a].r = seeds_point[j].r;
      pixels[a].g = seeds_point[j].g;
      pixels[a].b = seeds_point[j].b;
    }

    if(distance <= pixels[a].shortest_distance){
      pixels[a].shortest_distance = distance;

      pixels[a].r = seeds_point[j].r;
      pixels[a].g = seeds_point[j].g;
      pixels[a].b = seeds_point[j].b;

    }}}

   
// for (int i = 0; i < max_rows * max_col; i++)
// {
//   printf("x = %d, y = %d,  r = %d, g = %d, b = %d distance = %d\n", pixels[i].x, pixels[i].y, pixels[i].r, pixels[i].g, pixels[i].b, pixels[i].shortest_distance);
// }

 }



int main(){  
  int seeds;
  cout << "Type number of seeds: ";
  cin >> seeds;

  Point *pixels, *total_seed, *d_pixel, *d_total_seed;
  int *d_total_pixels;

    pixels = (Point*)malloc((max_rows *max_col)*sizeof(Point));
    total_seed = (Point*)malloc(seeds*sizeof(Point));

    hipMalloc(&d_pixel, (max_rows *max_col)*sizeof(Point));
    hipMalloc(&d_total_seed, (N)*sizeof(Point));
    hipMalloc(&d_total_pixels, (max_rows * max_col)*sizeof(int));
  for (int i = 0; i < 1; i++){
    for (int i = 0; i < seeds; i++){
      int x = rand() % max_rows;
      int y = rand() % max_col;
      int r = rand() % max_color;
      int g = rand() % max_color;
      int b = rand() % max_color;
     total_seed[i] = (Point(x, y, r, g, b));
      printf("x= %d, y = %d, r = %d, g = %d, b = %d \n",x , y,r, g, b);
    } }

long count = 0;
  for (int col = 0; col < max_col; col++){
      for (int rows = 0; rows < max_rows; rows++){
        pixels[count] = (Point(col, rows, 0, 0, 0));
        count++;
      } }
    
  //  int value = max_rows * max_col;
    hipMemcpy(d_pixel, pixels, sizeof(Point)*  (max_rows *max_col), hipMemcpyHostToDevice);
    hipMemcpy(d_total_seed, total_seed, sizeof(Point)* seeds, hipMemcpyHostToDevice);
    hipMemcpy(d_total_pixels, 0, sizeof(int)*  (max_rows *max_col), hipMemcpyHostToDevice);

  calculate_distanace_from_pixels_to_seeds<<<1, 1000>>>(d_pixel, d_total_seed, seeds, max_rows, max_col, d_total_pixels);

  
        hipFree(d_pixel);
        hipFree(d_total_seed);
        free(pixels);
        free(total_seed);
  return 0;
}

